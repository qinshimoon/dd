#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"

// Add uniaxial magnetocrystalline anisotropy field to B.
// http://www.southampton.ac.uk/~fangohr/software/oxs_uniaxial4.html
extern "C" __global__ void
adduniaxialanisotropy(float* __restrict__  Bx, float* __restrict__  By, float* __restrict__  Bz,
                      float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
                      float* __restrict__ K1LUT, float* __restrict__ K2LUT,
                      float* __restrict__ uxLUT, float* __restrict__ uyLUT, float* __restrict__ uzLUT,
                      uint8_t* __restrict__ regions, int N) {

	int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if (i < N) {

		uint8_t reg = regions[i];
		float  ux  = uxLUT[reg];
		float  uy  = uyLUT[reg];
		float  uz  = uzLUT[reg];
		float3 u   = normalized(make_float3(ux, uy, uz));
		float  K1  = K1LUT[reg];
		float  K2  = K2LUT[reg];
		float3 m   = {mx[i], my[i], mz[i]};
		float  mu  = dot(m, u);
		float3 Ba  = 2.0f*K1*    (mu)*u+ 
                     4.0f*K2*pow3(mu)*u;

		Bx[i] += Ba.x;
		By[i] += Ba.y;
		Bz[i] += Ba.z;
	}
}

